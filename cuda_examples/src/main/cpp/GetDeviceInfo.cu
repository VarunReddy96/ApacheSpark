// For infor about these example, see the links below
// https://devblogs.nvidia.com/how-query-device-properties-and-handle-errors-cuda-cc/
// https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__DEVICE.html#group__CUDART__DEVICE_1g1bf9d625a931d657e08db2b4391170f0
#include <iostream>

// Host code
int main(void)
{
    int nDevices;
    hipError_t err = hipGetDeviceCount(&nDevices);
    if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));

    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Max Threads Per Block: %d\n", prop.maxThreadsPerBlock);
        printf("  Max Threads Dim: x:%d, y:%d, z:%d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("  Max Grid Size: x:%d, y:%d, z:%d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("  Clock Rate (KHz): %d\n", prop.clockRate);
        printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }

}