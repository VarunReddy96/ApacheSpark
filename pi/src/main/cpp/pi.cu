/*
* A toy example of Pi estimation in CUDA
* A better version is here, https://docs.nvidia.com/cuda/curand/device-api-overview.html#thrust-and-curand-example
*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <math.h>
#include <sstream>
#include <sys/time.h>
#include <hiprand/hiprand_kernel.h>

// Accessible by ALL CPU and GPU functions !!!
__managed__ int points_in_circle;
__managed__ double rnd_seed;

// Kernel function
__global__
void countPoints(int total_points, float radius)
{
    // index = block index * number of threads per block + thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // stride  = number threads per block * number of block per grid
    int stride = blockDim.x * gridDim.x;

    hiprandState s;

    float r1, r2, x, y;
    float circle_x=radius, circle_y=radius;
    for (unsigned int i = index; i < total_points; i += stride) {
        // seed a random number generator
        hiprand_init(i+rnd_seed, 0, 0, &s);
        r1 = hiprand_uniform(&s);
        x = r1 * 2 * radius;
        r2 = hiprand_uniform(&s);
        y = r2 * 2 * radius;

        // Compare radius of circle with distance of its center from given point
        if ((x - circle_x) * (x - circle_x) +
                (y - circle_y) * (y - circle_y) <= radius * radius)
            atomicAdd(&points_in_circle, 1);
    }
}



// host code
int main(int argc, char* argv[]) {
  // Check the number of parameters
  if (argc < 3) {
      // Tell the user how to run the program
      std::cerr << "Usage: " << argv[0] << " TotalPoints Radius" << std::endl;
      return 1;
  }

  int total_points = std::stoi(argv[1]);
  std::string::size_type sz;     // alias of size_t
  float radius = std::stof(argv[2],&sz);

  // use the current time to create a seed for random number generator
  struct timeval t;
  gettimeofday(&t, 0);
  rnd_seed = 1000000.0*t.tv_sec/1000.0;

  points_in_circle=0;

  // Run kernel on some number of points on the GPU
  int blockSize = 256;
  int numBlocks = (total_points + blockSize - 1) / blockSize;
  countPoints<<<numBlocks, blockSize>>>(total_points, radius);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  float pi = 4.0f * points_in_circle / total_points;

  std::cout << std::setprecision(9);
  std::cout << "PI(" << points_in_circle << "/" << total_points << ")= "<< pi << std::endl;

  return 0;
}